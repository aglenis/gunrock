// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

//#include <gunrock/app/pbfs/pbfs_enactor.cuh>

// Operator includes
#include <gunrock/oprtr/edge_map_forward/kernel.cuh>
#include <gunrock/oprtr/vertex_map/kernel.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_bfs <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--instrumented] [--src=<source index>] [--quick] "
        "[--mark-pred] [--queue-sizing=<scale factor>]\n"
        "[--v]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --undirected If set then treat the graph as undirected.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --src Begins BFS from the vertex <source index>. If set as randomize\n"
        "  then will begin with a random source vertex.\n"
        "  If set as largestdegree then will begin with the node which has\n"
        "  largest degree.\n"
        "  --quick If set will skip the CPU validation code.\n"
        "  --mark-pred If set then keep not only label info but also predecessor info.\n"
        "  --queue-sizing Allocates a frontier queue sized at (graph-edges * <scale factor>).\n"
        "  Default is 1.0\n"
        );
 }

 /**
  * @brief Displays the BFS result (i.e., distance from source)
  *
  * @param[in] source_path Search depth from the source for each node.
  * @param[in] preds Predecessor node id for each node.
  * @param[in] nodes Number of nodes in the graph.
  * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
  */
 template<typename VertexId, typename SizeT>
 void DisplaySolution(VertexId *source_path, VertexId *preds, SizeT nodes, bool MARK_PREDECESSORS)
 {
    if (nodes > 40)
        nodes = 40;
    printf("[");
    for (VertexId i = 0; i < nodes; ++i) {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        printf(",");
        if (MARK_PREDECESSORS)
            PrintValue(preds[i]);
        printf(" ");
    }
    printf("]\n");
 }

 /**
  * Performance/Evaluation statistics
  */ 

struct Stats {
    char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * 
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where BFS starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the BFS algorithm
 * @param[in] total_queued Total element queued in BFS kernel running process
 * @param[in] avg_duty Average duty of the BFS kernels
 */
template<
    bool MARK_PREDECESSORS,
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    VertexId            *h_labels,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph.nodes; ++i) {
        if (h_labels[i] > -1) {
            ++nodes_visited;
            edges_visited += graph.row_offsets[i+1] - graph.row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0) {
        redundant_work = ((double) total_queued - edges_visited) / edges_visited;        // measure duplicate edges put through queue
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display statistics
    if (nodes_visited < 5) {
        printf("Fewer than 5 vertices visited.\n");
    } else {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf(" elapsed: %.3f ms, rate: %.3f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0) printf(", search_depth: %lld", (long long) search_depth);
        if (avg_duty != 0) {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        printf("\n src: %lld, nodes_visited: %lld, edges visited: %lld",
            (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (total_queued > 0) {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0) {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }
    
}




/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

 /**
  * @brief A simple CPU-based reference BFS ranking implementation.
  *
  * @tparam VertexId
  * @tparam Value
  * @tparam SizeT
  *
  * @param[in] graph Reference to the CSR graph we process on
  * @param[in] source_path Host-side vector to store CPU computed labels for each node
  * @param[in] src Source node where BFS starts
  */
 template<
    typename VertexId,
    typename Value,
    typename SizeT>
void SimpleReferenceBfs(
    const Csr<VertexId, Value, SizeT>       &graph,
    VertexId                                *source_path,
    VertexId                                src)
{
    //initialize distances
    for (VertexId i = 0; i < graph.nodes; ++i) {
        source_path[i] = -1;
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    //
    //Perform BFS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty()) {
        
        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;

        // Locate adjacency list
        int edges_begin = graph.row_offsets[dequeued_node];
        int edges_end = graph.row_offsets[dequeued_node + 1];

        for (int edge = edges_begin; edge < edges_end; ++edge) {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = graph.column_indices[edge];
            if (source_path[neighbor] == -1) {
                source_path[neighbor] = neighbor_dist;
                if (search_depth < neighbor_dist) {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU BFS finished in %lf msec. Search depth is:%d\n", elapsed, search_depth);
}

/**
 * @brief Run BFS tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node where BFS starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] max_queue_sizing Scaling factor used in edge mapping
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    VertexId src,
    int max_grid_size,
    int num_gpus,
    double max_queue_sizing,
    std::string partition_method,
    int*  gpu_idx)
{
    
    typedef BFSProblem<
        VertexId,
        SizeT,
        Value,
        MARK_PREDECESSORS,
        ENABLE_IDEMPOTENCE,
        (MARK_PREDECESSORS && ENABLE_IDEMPOTENCE)> Problem; // does not use double buffer

        // Allocate host-side label array (for both reference and gpu-computed results)
        VertexId    *reference_labels       = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
        VertexId    *h_labels               = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
        VertexId    *reference_check        = (g_quick) ? NULL : reference_labels;
        VertexId    *h_preds                = NULL;
        if (MARK_PREDECESSORS) {
            h_preds = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
        }

        printf("RunTests begin.\n"); fflush(stdout);
        // Allocate BFS enactor map
        BFSEnactor<INSTRUMENT>* bfs_enactor=new BFSEnactor<INSTRUMENT>(g_verbose);
        printf("bfs_enactor created.\n"); fflush(stdout);

        // Allocate problem on GPU
        Problem *csr_problem = new Problem;
        printf("problem created.\n"); fflush(stdout);

        util::GRError(csr_problem->Init(
            g_stream_from_host,
            partition_method,
            graph,
            num_gpus,gpu_idx), "Problem BFS Initialization Failed", __FILE__, __LINE__);
        printf("problem inited.\n"); fflush(stdout);

        //
        // Compute reference CPU BFS solution for source-distance
        //
        if (reference_check != NULL)
        {
            printf("compute ref value\n");
            SimpleReferenceBfs(
                    graph,
                    reference_check,
                    src);
            printf("\n");
        }

        Stats *stats = new Stats("GPU BFS");

        long long           total_queued = 0;
        VertexId            search_depth = 0;
        double              avg_duty = 0.0;

        // Perform BFS
        GpuTimer gpu_timer;

        util::GRError(csr_problem->Reset(src, bfs_enactor->GetFrontierType(), max_queue_sizing), "BFS Problem Data Reset Failed", __FILE__, __LINE__);
        printf("BFSProblem reseted.\n"); fflush(stdout);
        gpu_timer.Start();
        util::GRError(bfs_enactor->template Enact<Problem>(csr_problem, src, max_grid_size), "BFS Problem Enact Failed", __FILE__, __LINE__);
        gpu_timer.Stop();

        bfs_enactor->GetStatistics(total_queued, search_depth, avg_duty);

        float elapsed = gpu_timer.ElapsedMillis();

        // Copy out results
        util::GRError(csr_problem->Extract(h_labels, h_preds), "BFS Problem Data Extraction Failed", __FILE__, __LINE__);

        // Verify the result
        if (reference_check != NULL) {
            printf("Validity: ");
            CompareResults(h_labels, reference_check, graph.nodes, true);
        }
        printf("\nFirst 40 labels of the GPU result."); 
        // Display Solution
        DisplaySolution(h_labels, h_preds, graph.nodes, MARK_PREDECESSORS);

        DisplayStats<MARK_PREDECESSORS>(
            *stats,
            src,
            h_labels,
            graph,
            elapsed,
            search_depth,
            total_queued,
            avg_duty);


        // Cleanup
        delete stats;
        if (bfs_enactor)      {delete bfs_enactor;    bfs_enactor     =NULL;printf("bfs_enactor deleted.\n"   );fflush(stdout);}
        if (csr_problem)      {delete csr_problem;    csr_problem     =NULL;printf("csr_problem deleted.\n"   );fflush(stdout);}
        if (reference_labels) {free(reference_labels);reference_labels=NULL;printf("reference_labels freed.\n");fflush(stdout);}
        if (h_labels)         {free(h_labels        );h_labels        =NULL;printf("h_labels freed.\n"        );fflush(stdout);}
        if (h_preds)          {free(h_preds         );h_preds         =NULL;printf("h_preds freed.\n"         );fflush(stdout);}

        hipDeviceSynchronize();
        printf("RunTests end.\n"); fflush(stdout);
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs &args)
{
    VertexId            src                 = -1;           // Use whatever the specified graph-type's default is
    std::string         src_str;
    bool                instrumented        = false;        // Whether or not to collect instrumentation from kernels
    bool                mark_pred           = false;        // Whether or not to mark src-distance vs. parent vertices
    bool                idempotence         = false;        // Whether or not to enable idempotence operation
    int                 max_grid_size       = 0;            // maximum grid size (0: leave it up to the enactor)
    int                 num_gpus            = 1;            // Number of GPUs for multi-gpu enactor to use
    double              max_queue_sizing    = 1.0;          // Maximum size scaling factor for work queues (e.g., 1.0 creates n and m-element vertex and edge frontiers).
    std::string         partition_method    = "random";
    int*                gpu_idx             = NULL;

    printf("RunTests begin.\n");fflush(stdout);

    instrumented = args.CheckCmdLineFlag("instrumented");
    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty()) {
        src = 0;
    } else if (src_str.compare("randomize") == 0) {
        src = graphio::RandomNode(graph.nodes);
    } else if (src_str.compare("largestdegree") == 0) {
        src = graph.GetNodeWithHighestDegree();
    } else {
        args.GetCmdLineArgument("src", src);
    }

    //printf("Display neighbor list of src:\n");
    //graph.DisplayNeighborList(src);

    g_quick = args.CheckCmdLineFlag("quick");
    mark_pred = args.CheckCmdLineFlag("mark-pred");
    idempotence = args.CheckCmdLineFlag("idempotence");
    args.GetCmdLineArgument("queue-sizing", max_queue_sizing);
    g_verbose = args.CheckCmdLineFlag("v");
    if (args.CheckCmdLineFlag("partition_method")) args.GetCmdLineArgument("partition_method",partition_method);
    if (args.CheckCmdLineFlag("device"))
    {
        std::vector<int> gpus;
        args.GetCmdLineArguments<int>("device",gpus);
        num_gpus   = gpus.size();
        printf("Using %d gpus: ", num_gpus);
        gpu_idx    = new int[num_gpus];
        for (int i=0;i<num_gpus;i++) {gpu_idx[i]=gpus[i]; printf(" %d ", gpu_idx[i]);}
        printf("\n"); fflush(stdout);
    } else {
        num_gpus   = 1;
        gpu_idx    = new int[1];
        gpu_idx[0] = -1;
    }

    printf("RunTests cmdLine reading finished.\n");fflush(stdout);
    if (instrumented) {
        if (mark_pred) {
            if (idempotence) {
                RunTests<VertexId, Value, SizeT, true, true, true>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        partition_method,
                        gpu_idx);
            } else {
                RunTests<VertexId, Value, SizeT, true, true, false>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        partition_method,
                        gpu_idx);
            }
        } else {
            if (idempotence) {
                RunTests<VertexId, Value, SizeT, true, false, true>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        partition_method,
                        gpu_idx);
            } else {
                RunTests<VertexId, Value, SizeT, true, false, false>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        partition_method,
                        gpu_idx);
            }
        }
    } else {
        if (mark_pred) {
            if (idempotence) {
                RunTests<VertexId, Value, SizeT, false, true, true>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        partition_method,
                        gpu_idx);
            } else {
                RunTests<VertexId, Value, SizeT, false, true, false>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        partition_method,
                        gpu_idx);
            }
        } else {
            if (idempotence) {
                RunTests<VertexId, Value, SizeT, false, false, true>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        partition_method,
                        gpu_idx);
            } else {
                RunTests<VertexId, Value, SizeT, false, false, false>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        partition_method,
                        gpu_idx);
            }
        }
    }
    delete[] gpu_idx;gpu_idx=NULL;
    printf("RunTests returned.\n"); fflush(stdout);
}



/******************************************************************************
* Main
******************************************************************************/

int main( int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
		Usage();
		return 1;
	}

	DeviceInit(args);
	hipSetDeviceFlags(hipDeviceMapHost);

	//srand(0);									// Presently deterministic
	//srand(time(NULL));

	// Parse graph-contruction params
	g_undirected = args.CheckCmdLineFlag("undirected");

	std::string graph_type = argv[1];
	int flags = args.ParsedArgc();
	int graph_args = argc - flags - 1;

	if (graph_args < 1) {
		Usage();
		return 1;
	}
	
	//
	// Construct graph and perform search(es)
	//

	if (graph_type == "market") {

		// Matrix-market coordinate-formatted graph file

		typedef int VertexId;							// Use as the node identifier type
		typedef int Value;								// Use as the value type
		typedef int SizeT;								// Use as the graph size type
		Csr<VertexId, Value, SizeT> csr(false);         // default value for stream_from_host is false

		if (graph_args < 1) { Usage(); return 1; }
		char *market_filename = (graph_args == 2) ? argv[2] : NULL;
		if (graphio::BuildMarketGraph<false>(
			market_filename, 
			csr, 
			g_undirected,
			false) != 0) // no inverse graph
		{
			return 1;
		}

		csr.PrintHistogram();

		// Run tests
		RunTests(csr, args);

	} else {

		// Unknown graph type
		fprintf(stderr, "Unspecified graph type\n");
		return 1;

	}

        printf("Program ending.\n"); fflush(stdout);
	return 0;
}
